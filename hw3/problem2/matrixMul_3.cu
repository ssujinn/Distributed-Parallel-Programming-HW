
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define R 4096
#define C 4096
#define BLOCK_W 32
#define BLOCK_H 32
#define SHARED_SIZE (BLOCK_W*BLOCK_H*4)
#define NSTREAM 1

float *a, *b, *result;

__global__ void multiple(float *A, float *B, float *res){
    __shared__ float shared_A[BLOCK_W * BLOCK_H];
    __shared__ float shared_B[BLOCK_W * BLOCK_H];
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float ret = 0;

    for (int k = 0; k < (C - 1) / BLOCK_W + 1; k++){
        if (row < R && k * BLOCK_W + tx < C)
            shared_A[ty * BLOCK_W + tx] = A[row * C + k * BLOCK_W + tx];
        else
            shared_A[ty * BLOCK_W + tx] = 0;

        if (col < C && k * BLOCK_H + ty < R)
            shared_B[ty * BLOCK_W + tx] = B[(k * BLOCK_W + ty) * C + col];
        else
            shared_B[ty * BLOCK_W + tx] = 0;

        __syncthreads();

        for (int i = 0; i < BLOCK_W; i++){
          ret += shared_A[ty * BLOCK_W + i] * shared_B[i * BLOCK_W + tx];
        }
        __syncthreads();
    }

    if (row < C && col < C)
        res[row * C + col] = ret;
}

int main(){
  float *d_a, *d_b, *d_r;
  hipEvent_t start, end;
  float etime;
  dim3 block(BLOCK_W, BLOCK_H);
  dim3 grid(C/BLOCK_W, R/BLOCK_H);

  hipStream_t stream[NSTREAM];
  for (int i = 0 ; i < NSTREAM; i++)
    hipStreamCreate(&stream[i]);

  a = (float*)malloc(sizeof(float)*R*C);
  b = (float*)malloc(sizeof(float)*R*C);
  result = (float*)malloc(sizeof(float)*R*C);

  hipEventCreate(&start);
	hipEventCreate(&end);
	srand(time(NULL));

  // random number creation
	for (int i = 0; i < R; i++)
    for (int j = 0; j < C; j++){
       a[i * C + j] = (rand() % 1000000) / (float)10000;
       b[i * C + j] = (rand() % 1000000) / (float)10000;
    }

  // cuda var initialization
  hipMalloc((void**)&d_a, sizeof(float)*R*C);
  hipMalloc((void**)&d_b, sizeof(float)*R*C);
  hipMalloc((void**)&d_r, sizeof(float)*R*C);

    // kernel call & exec time check
  hipEventRecord(start, 0);
  hipMemcpy(d_b, b, sizeof(float)*R*C, hipMemcpyHostToDevice);
  for (int i = 0; i < NSTREAM; i++){
      int ELES_PER_STREAM = R * C / NSTREAM;
      int offset = i * ELES_PER_STREAM;
      hipMemcpyAsync(&d_a[offset], &a[offset], sizeof(float)*ELES_PER_STREAM, hipMemcpyHostToDevice, stream[i]);
      multiple<<<grid, block, 2 * SHARED_SIZE, stream[i]>>>(d_a + offset, d_b, d_r + offset);
      hipMemcpyAsync(&result[offset], &d_r[offset], sizeof(float)*ELES_PER_STREAM, hipMemcpyDeviceToHost, stream[i]);
  }

  for (int i = 0; i < NSTREAM; i++)
    hipStreamSynchronize(stream[i]);
  
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&etime, start, end);

	printf("EXEC TIME : %f ms\n", etime);

  hipEventDestroy(start);
	hipEventDestroy(end);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_r);

	return 0;
}