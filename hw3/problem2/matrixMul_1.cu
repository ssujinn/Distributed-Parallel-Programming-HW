
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define R 4096
#define C 4096
#define BLOCK_W 32
#define BLOCK_H 32

float *a, *b, *result;

__global__ void multiple(float *A, float *B, float *res){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float ret = 0;

    for (int i = 0; i < C; i += BLOCK_W){
        for (int j = i; j < i + BLOCK_W; j++){
            ret += A[row * C + j] * B[j * C + col];
        }
    }

    res[row * C + col] = ret;
}

int main(){
  float *d_a, *d_b, *d_r;
  hipEvent_t start, end;
  float etime;
  dim3 block(BLOCK_W, BLOCK_H);
  dim3 grid(C/BLOCK_W, R/BLOCK_H);

  a = (float*)malloc(sizeof(float)*R*C);
  b = (float*)malloc(sizeof(float)*R*C);
  result = (float*)malloc(sizeof(float)*R*C);

  hipEventCreate(&start);
	hipEventCreate(&end);
	srand(time(NULL));

  // random number creation
	for (int i = 0; i < R; i++)
    for (int j = 0; j < C; j++){
       a[i * C + j] = (rand() % 1000000) / (float)10000;
       b[i * C + j] = (rand() % 1000000) / (float)10000;
    }

  // cuda var initialization
  hipMalloc((void**)&d_a, sizeof(float)*R*C);
  hipMalloc((void**)&d_b, sizeof(float)*R*C);
  hipMalloc((void**)&d_r, sizeof(float)*R*C);
  

  // kernel call & exec time check
  hipEventRecord(start, 0);
  hipMemcpy(d_a, a, sizeof(float)*R*C, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float)*R*C, hipMemcpyHostToDevice);
	multiple<<<grid, block>>>(d_a, d_b, d_r);
  hipMemcpy(result, d_r, sizeof(float)*R*C, hipMemcpyDeviceToHost);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&etime, start, end);

	printf("EXEC TIME : %f ms\n", etime);

  hipEventDestroy(start);
	hipEventDestroy(end);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_r);

	return 0;
}