
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_ELEMENT 100000
#define BLOCK_SIZE 32
#define GRID_SIZE ((NUM_ELEMENT + BLOCK_SIZE - 1) / BLOCK_SIZE)
#define WARP_SIZE 32
#define SHARED_SIZE (BLOCK_SIZE * 4)

__global__ void reduction_max(int *Arr, int *Max){
    extern __shared__ int sharedmem[];
    int tid = threadIdx.x;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    sharedmem[tid] = Arr[id];
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2){
        if ((tid / WARP_SIZE) % (2*i) == 0){
            if (sharedmem[tid] < sharedmem[tid + i])
              sharedmem[tid] = sharedmem[tid + i];
        }
        __syncthreads();
    }

    if (tid == 0)
      Max[0] = sharedmem[0];
}

int main(){
	int* arr;
  int* d_arr, *d_max;
	int max = 0;
  hipEvent_t start, end;
  float etime;
  dim3 block(BLOCK_SIZE);
  dim3 grid(GRID_SIZE);

  hipEventCreate(&start);
	hipEventCreate(&end);
	srand(time(NULL));

  // random number creation
	arr = (int*)malloc(sizeof(int) * NUM_ELEMENT);
	for (int i = 0; i < NUM_ELEMENT; i++)
		arr[i] = rand() % (NUM_ELEMENT * 10);

  // tmp print
	//for (int i = 0; i < NUM_ELEMENT; i++)
	//	printf("%d\n", arr[i]);

  // cuda var initialization
  hipMalloc((void**)&d_arr, sizeof(int)*NUM_ELEMENT);
  hipMalloc((void**)&d_max, sizeof(int));
  hipMemcpy(d_arr, arr, sizeof(int)*NUM_ELEMENT, hipMemcpyHostToDevice);

  // kernel call & exec time check
  hipEventRecord(start, 0);
	reduction_max<<<grid, block, SHARED_SIZE>>>(d_arr, d_max);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&etime, start, end);

  hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

	printf("MAX NUM : %d\n", max);
	printf("EXEC TIME : %f ms\n", etime);

  hipEventDestroy(start);
	hipEventDestroy(end);
  hipFree(d_arr);
  hipFree(d_max);

	return 0;
}