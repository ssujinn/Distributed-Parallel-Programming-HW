
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_ELEMENT 100000
#define BLOCK_SIZE 128
#define GRID_SIZE ((NUM_ELEMENT + BLOCK_SIZE - 1) / BLOCK_SIZE)
#define WARP_SIZE 32

__global__ void reduction_max(int *Arr, int *Max){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 1; i < blockDim.x; i *= 2){
        if ((id / WARP_SIZE) % (2*i) == 0){
            if (Arr[id] < Arr[id + i])
              Arr[id] = Arr[id + i];
        }
        __syncthreads();
    }

    if (id == 0)
      Max[0] = Arr[0];
}

int main(){
	int* arr;
  int* d_arr, *d_max;
	int max = 0;
  hipEvent_t start, end;
  float etime;
  dim3 block(BLOCK_SIZE);
  dim3 grid(GRID_SIZE);

  hipEventCreate(&start);
	hipEventCreate(&end);
	srand(time(NULL));

  // random number creation
	arr = (int*)malloc(sizeof(int) * NUM_ELEMENT);
	for (int i = 0; i < NUM_ELEMENT; i++)
		arr[i] = rand() % (NUM_ELEMENT * 10);

  // tmp print
	//for (int i = 0; i < NUM_ELEMENT; i++)
	//	printf("%d\n", arr[i]);

  // cuda var initialization
  hipMalloc((void**)&d_arr, sizeof(int)*NUM_ELEMENT);
  hipMalloc((void**)&d_max, sizeof(int));
  hipMemcpy(d_arr, arr, sizeof(int)*NUM_ELEMENT, hipMemcpyHostToDevice);

  // kernel call & exec time check
  hipEventRecord(start, 0);
	reduction_max<<<grid, block>>>(d_arr, d_max);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&etime, start, end);

  hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

	printf("MAX NUM : %d\n", max);
	printf("EXEC TIME : %f ms\n", etime);

  hipEventDestroy(start);
	hipEventDestroy(end);
  hipFree(d_arr);
  hipFree(d_max);

	return 0;
}